/*
 * This program uses the device CURAND API to calculate what 
 * proportion of pseudo-random ints have low bit set.
 * It then generates uniform results to calculate how many
 * are greater than .5.
 * It then generates  normal results to calculate how many 
 * are within one standard deviation of the mean.
 */
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * 64;
    /* Each thread gets same seed, a different sequence 
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
    //curand_init(id, 0, 0, &state[id]);
}

__global__ void generate_normal_kernel(hiprandState *state,
                                int n, 
                                float *result)
{
    int id = threadIdx.x + blockIdx.x * 64;
    float x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random normals */
    for(int i = 0; i < n; i++) {
        x = hiprand_normal(&localState);
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] = x;
}


int main(int argc, char *argv[])
{

    int i;
    unsigned int total;
    hiprandState *devStates;
    //unsigned int *devResults, *hostResults;
    float *devResults, *hostResults;
    bool useMRG = 0;
    bool usePHILOX = 0;
    int sampleCount = 10000;
    bool doubleSupported = 0;
    int device;
    struct hipDeviceProp_t properties;

    /* check for double precision support */
    CUDA_CALL(hipGetDevice(&device));
    CUDA_CALL(hipGetDeviceProperties(&properties,device));
    if ( properties.major >= 2 || (properties.major == 1 && properties.minor >= 3) ) {
        doubleSupported = 1;
    }
    
    /* Allocate space for results on host */
    hostResults = (float *)calloc(64 * 64, sizeof(float));

    /* Allocate space for results on device */
    CUDA_CALL(hipMalloc((void **)&devResults, 64 * 64 * 
              sizeof(float)));

    /* Set results to 0 */
    CUDA_CALL(hipMemset(devResults, 0, 64 * 64 * 
              sizeof(float)));

    /* Allocate space for prng states on device */
    CUDA_CALL(hipMalloc((void **)&devStates, 64 * 64 * 
                  sizeof(hiprandState)));
    
    /* Setup prng states */
    setup_kernel<<<64, 64>>>(devStates);
    
    /* Set results to 0 */
    CUDA_CALL(hipMemset(devResults, 0, 64 * 64 * 
              sizeof(float)));

    /* Generate and use normal pseudo-random  */
    for(i = 0; i < 50; i++) {
      generate_normal_kernel<<<64, 64>>>(devStates, sampleCount, devResults);
    }

    /* Copy device memory to host */
    CUDA_CALL(hipMemcpy(hostResults, devResults, 64 * 64 * 
        sizeof(float), hipMemcpyDeviceToHost));

    /* Show result */
    total = 0;
    for(i = 0; i < 64 * 64; i++) {
      printf("%lf\n",hostResults[i]);
    }

    /* Cleanup */
    CUDA_CALL(hipFree(devStates));
    CUDA_CALL(hipFree(devResults));
    free(hostResults);
    return EXIT_SUCCESS;
}
